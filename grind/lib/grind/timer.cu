/*
 * Copyright (c) 2009 Dr. D Studios. (Please refer to license for details)
 * SVN_META_HEADURL = "$HeadURL: $"
 * SVN_META_ID = "$Id: $"
 */

//-------------------------------------------------------------------------------------------------
#include "timer.h"
#include "hip/hip_runtime.h"

namespace grind
{

	GPUTimer::GPUTimer()
	{
	  e_start = new hipEvent_t;
	  e_stop = new hipEvent_t;

	  hipEventCreate((hipEvent_t *)e_start);
	  hipEventCreate((hipEvent_t *)e_stop);
	}

	GPUTimer::~GPUTimer()
	{
	  hipEventDestroy(*((hipEvent_t *)e_start));
	  hipEventDestroy(*((hipEvent_t *)e_stop));

	  delete (hipEvent_t *)e_start;
	  delete (hipEvent_t *)e_stop;
	}

	void GPUTimer::start() {
	  hipEventRecord(*((hipEvent_t *)e_start), 0);
	}

	void GPUTimer::stop()  {
	  hipEventRecord(*((hipEvent_t *)e_stop), 0);
	}

	float GPUTimer::elapsed_ms()
	{
	    hipEventSynchronize(*((hipEvent_t *)e_stop));
	    float ms;
	    hipEventElapsedTime(&ms, *((hipEvent_t *)e_start), *((hipEvent_t *)e_stop));
	    return ms;
	}

} // grind
